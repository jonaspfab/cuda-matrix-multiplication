#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "NaiveMatrixMulti.cu"
#include "TilingMatrixMulti.cu"

using namespace std;

#define NAIVE_1D 1
#define NAIVE_2D 2
#define TILING 3
#define TILING_LOOP_UNROLLING 4
#define CU_BLAS 5

/**
 * Multiplies matrix 'A' and 'B' and stores result in 'Y' using the cuBLAS
 * library
 *
 * Note that the matrix parameters must reference the device memory
 *
 * @param handle Created CuBlas handle
 */
void cuBlasMM(hipblasHandle_t handle, int n, double *d_A, double *d_B, double *d_Y) {
    double alpha = 1.0;
	double beta = 0.0;

	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_B, n, d_A, n, &beta, d_Y, n);
    hipDeviceSynchronize();

    hipblasDestroy(handle);
}

/** Checks if matrix 'A' multiplied with matrix 'B' is 'Y' */
bool isResultCorrect(int n, double *A, double *B, double *Y) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
            double result = 0;
			for (int k = 0; k < n; k++)
				result += A[i * n + k] * B[k * n + j];

            if (abs(result - Y[i * n + j]) > 0.01)
                return false;
		}
	}

    return true;
}

/** Fills given matrix 'M' with random numbers */
void fillMatrix(int n, double *M) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++)
            M[j + i * n] = (double) ((int) rand() % 10);
	}
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        cout << "usage: " << argv[0] << " n strategy [-v]" << endl;
        cout << endl << "positional arguments:" << endl;
        cout << " n         Matrix size (Must be multiple of 32 for strategy 3 and 4)" << endl;
        cout << " strategy  Defines matrix multiplication strategy" << endl;
        cout << "            - 1 for naive matrix multi with 1D blocks and grid" << endl;
        cout << "            - 2 for naive matrix multi with 2D blocks and grid" << endl;
        cout << "            - 3 for tiling matrix multi" << endl;
        cout << "            - 4 for for tiling matrix multi with loop unrolling" << endl;
        cout << endl << "optional arguments:" << endl;
        cout << " -v        Validate matrix multiplication result" << endl;

        return -1;
    }

    // Parse command line arguments
	int n = atoi(argv[1]);
    int strategy = atoi(argv[2]);
    bool validateResult = argc > 3 ? !strcmp(argv[3], "-v") : false;

    // Create handle here so it won't be included in timing
    hipblasHandle_t handle;
    if (strategy == CU_BLAS)
	    hipblasCreate(&handle);

    // Matrices stored on host memory
    double *A, *B, *Y;
    // Matrices stored on device memory
    double *d_A, *d_B, *d_Y;
    int size = n * n * sizeof(double);

	A = (double *)malloc(size);
	B = (double *)malloc(size);
	Y = (double *)malloc(size);

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_Y, size);

    // Fill matrices with random values
    fillMatrix(n, A);
    fillMatrix(n, B);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    clock_t t = clock();

    switch (strategy) {
        case NAIVE_1D:
            naiveMM(n, d_A, d_B, d_Y, false);
            break;
        case NAIVE_2D:
            naiveMM(n, d_A, d_B, d_Y, true);
            break;
        case TILING:
            tilingMM(n, d_A, d_B, d_Y, false);
            break;
        case TILING_LOOP_UNROLLING:
            tilingMM(n, d_A, d_B, d_Y, true);
            break;
        case CU_BLAS:
            cuBlasMM(handle, n, d_A, d_B, d_Y);
            break;
        default:
            cout << "\'" << strategy << "\' is not a valid strategy" << endl;
            return -1;
    }

    t = clock() - t;

    hipMemcpy(Y, d_Y, size, hipMemcpyDeviceToHost);

    double nD = (double) n;

    cout << n << "\t";
    cout << ((nD / 1000000.0) * nD * nD) / (t / ((double) CLOCKS_PER_SEC)) << endl;
    if (validateResult)
        cout << "Result is " << (isResultCorrect(n, A, B, Y) ? "correct" : "incorrect") << endl;

    free(A); free(B); free(Y);
    hipFree(d_A); hipFree(d_B); hipFree(d_Y);

    return 0;
}
